#include "hip/hip_runtime.h"
﻿#include "WaveEquationKernel.cuh"

#include "Surface.h"
#include <stdio.h>
#include <cuda_gl_interop.h>


__global__ void JacobiKernel(float4 *u0, float4 *u1, int n, float a, float b)
{
	const int c = blockIdx.x * blockDim.x + threadIdx.x;
	const int r = blockIdx.y * blockDim.y + threadIdx.y;

	if((c >= n) || (r >= n)) return;
	const int i = c + r * n;	// 1D indexing.

	float ui_mw = 0;
	float ui_m1 = 0;
	float ui_p1 = 0;
	float ui_pw = 0;

	if(i - n >= 0)
		ui_mw = u1[i - n].y;
	if(i - 1 >= 0)
		ui_m1 = u1[i - 1].y;
	if(i + 1 < n * n)
		ui_p1 = u1[i + 1].y;
	if(i + n < n * n)
		ui_pw = u1[i + n].y;

	float ax = b * (ui_mw + ui_m1 + ui_p1 + ui_pw);
	float res = (2.0f * u1[i].y - u0[i].y - ax) / a;
	u0[i].y = u1[i].y;

	u1[i].y = res;

	// u1[i].y = u0[i].y + 0.05f;
	// u0[i].y = u1[i].y;
}


void kernelLauncher(float4 *u0, float4 *u1, int n, float a, float b)
{
#define TX 16
#define TY 16

	const dim3 blockSize(TX, TY);
	const dim3 gridSize = dim3((n + TX - 1)/TX, (n + TY - 1)/TY);
	JacobiKernel<<<gridSize, blockSize>>>(u0, u1, n, a, b);
}

//-----------------------

// GPU 내의 데이터에 바로 접근해서 계산을 하기 때문에
// wave buffer의 초기화가 끝난 후 실행이 되어야 한다.
hipError_t initCudaDevice()
{
	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if(cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}
	printf("hipSetDevice finished\n");

Error:
	return cudaStatus;
}