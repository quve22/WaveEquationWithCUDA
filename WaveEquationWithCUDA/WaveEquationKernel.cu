#include "hip/hip_runtime.h"
#include "WaveEquationKernel.cuh"

#include "Surface.h"
#include <stdio.h>
#include <cuda_gl_interop.h>

float4 *cuda_data = NULL;

extern "C" void map_texture(hipGraphicsResource *resource, int w, int h)
{
	size_t size;
	hipGraphicsResourceGetMappedPointer((void **)(&cuda_data), &size, resource);
}


hipGraphicsResource *resources[1];

__global__ void JacobiKernel()
{
	
}

// GPU ���� �����Ϳ� �ٷ� �����ؼ� ����� �ϱ� ������
// wave buffer�� �ʱ�ȭ�� ���� �� ������ �Ǿ�� �Ѵ�.
hipError_t initCudaDevice()
{
	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if(cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}
	printf("hipSetDevice finished\n");

Error:

	return cudaStatus;
}

void set_cuda_ogl_interoperability()
{
	GLuint pbo;

	glGenBuffers(1, &pbo);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo);
	glBufferData(GL_PIXEL_UNPACK_BUFFER, GRID * GRID * sizeof(float), NULL, GL_DYNAMIC_DRAW);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);

	cudaGLSetGLDevice(0);

	// Register Pixel Buffer Object as CUDA graphics resource.
	hipGraphicsGLRegisterBuffer(resources, pbo, cudaGraphicsMapFlagsNone);

	hipStream_t hip_stream;
	hipStreamCreate(&hip_stream);

	// �׷��� ���ҽ��� CUDA ��Ʈ���� ����.
	hipGraphicsMapResources(1, resources, hip_stream);

	// CUDA Function ȣ��.
	map_texture(resources[0], GRID, GRID);

	hipGraphicsUnmapResources(1, resources, hip_stream);
	hipStreamDestroy(hip_stream);
}

hipError_t calculateWaveEquation()
{
	dim3 blockSize(16, 16);
	dim3 gridSize(GRID / blockSize.x, GRID / blockSize.y);

	hipStream_t hip_stream;
	hipError_t cudaStatus;
	// printf("calculateWaveEquation() started\n");

	hipStreamCreate(&hip_stream);
	hipGraphicsMapResources(1, resources, hip_stream);

	JacobiKernel <<< 16, 16 >>>();

	cudaStatus = hipGetLastError();
	if(cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

Error:
	hipGraphicsUnmapResources(1, resources, hip_stream);
	hipStreamDestroy(hip_stream);
	// printf("calculateWaveEquation() finished\n");
	return cudaStatus;
}